#include <cstddef>
#include <hip/hip_runtime.h>
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime_api.h>
#include <hip/driver_types.h>
#include <iostream>
#include <math.h>
#include <string>
#include <vector_functions.h>

struct Particle {
  float x;
  float y;
  float z;

  float px;
  float py;
  float pz;

  // Default constructor
  Particle() : x(), y(), z(), px(), py(), pz() {}

  void update(float x_pos, float y_pos, float z_pos, float x_momentum,
              float y_momentum, float z_momentum) {
    x = x_pos;
    y = y_pos;
    z = z_pos;
    px = x_momentum;
    py = y_momentum;
    pz = z_momentum;
  }

  std::string print() {
    return "x: " + std::to_string(x) + " y: " + std::to_string(y) +
           " z: " + std::to_string(z) + " px: " + std::to_string(px) +
           " py: " + std::to_string(py) + " pz: " + std::to_string(pz);
  }
};

struct ParticleHistory {
  float *x;
  float *y;
  float *z;
  float *px;
  float *py;
  float *pz;

  // Default constructor
  ParticleHistory()
      : x(nullptr), y(nullptr), z(nullptr), px(nullptr), py(nullptr),
        pz(nullptr) {}

  ~ParticleHistory() {
    delete[] x;
    delete[] y;
    delete[] z;
    delete[] px;
    delete[] py;
    delete[] pz;
  }
};

enum class ParticlePlacementType {
  UNIFORM, // Evenly spaced integer placement for xyz coordinates
};

void initialize_particles(Particle *particles, float3 num_of_particles,
                          int total_particle_count, float3 system_length,
                          ParticlePlacementType placement_type) {
  switch (placement_type) {
  case ParticlePlacementType::UNIFORM:
    float deltax = system_length.x / num_of_particles.x;
    float deltay = system_length.y / num_of_particles.y;
    float deltaz = system_length.z / num_of_particles.z;
    for (int i = 0; i < total_particle_count; i++) {
      float corrected_index = static_cast<float>(i) + 0.5f;
      particles[i].update(deltax * corrected_index, deltay * corrected_index,
                          deltaz * corrected_index, 10, 10, 10);
    }
    break;
  }
}

void initialize_particle_history(ParticleHistory *particle_histories,
                                 int total_particle_count, int max_iter) {
  for (int i = 0; i < total_particle_count; i++) {
    particle_histories[i].x = new float[max_iter];
    particle_histories[i].y = new float[max_iter];
    particle_histories[i].z = new float[max_iter];
    particle_histories[i].px = new float[max_iter];
    particle_histories[i].py = new float[max_iter];
    particle_histories[i].pz = new float[max_iter];
  }
}

void initialize_particle_history_cuda(ParticleHistory *particle_histories,
                                 int total_particle_count, int max_iter) {
  for (int i = 0; i < total_particle_count; i++) {
    particle_histories[i].x = new float[max_iter];
    particle_histories[i].y = new float[max_iter];
    particle_histories[i].z = new float[max_iter];
    particle_histories[i].px = new float[max_iter];
    particle_histories[i].py = new float[max_iter];
    particle_histories[i].pz = new float[max_iter];
  }
}

__device__ void push_particle(Particle &particle, const float3 &e_field,
                              const float3 &b_field, const float timestep,
                              const float charge, float mass) {
  // Pre-compute reused calculations
  float mass_square = mass * mass;
  float timestep_charge = charge * timestep;

  // half-step momentum from electric field
  float px_half = particle.px + timestep_charge * e_field.x / 2.0f;
  float py_half = particle.py + timestep_charge * e_field.y / 2.0f;
  float pz_half = particle.pz + timestep_charge * e_field.z / 2.0f;

  // lorentz factor for half-step momentum
  float lorentz = sqrtf(1.0f + px_half * px_half + py_half * py_half +
                        pz_half * pz_half / mass_square);

  // rotation vector from magnetic field
  float lorentz_double = 2.0f * lorentz; // Pre-compute to avoid recalculating
  float tx = timestep_charge * b_field.x / lorentz_double;
  float ty = timestep_charge * b_field.y / lorentz_double;
  float tz = timestep_charge * b_field.z / lorentz_double;
  float t_mag_square = tx * tx + ty * ty + tz * tz;

  // cross product of half p and t
  float px_prime = px_half + (py_half * tz - pz_half * ty);
  float py_prime = py_half + (pz_half * tx - px_half * tz);
  float pz_prime = pz_half + (px_half * ty - py_half * tx);

  // update momentum with effect from boris rotation and electric field
  float denominator = 1 + t_mag_square;
  float px_updated = px_half +
                     2 * (py_prime * tz - pz_prime * ty) / denominator +
                     timestep_charge * e_field.x / 2.0f;
  float py_updated = py_half +
                     2 * (pz_prime * tx - px_prime * tz) / denominator +
                     timestep_charge * e_field.y / 2.0f;
  float pz_updated = pz_half +
                     2 * (px_prime * ty - py_prime * tx) / denominator +
                     timestep_charge * e_field.z / 2.0f;

  // lorentz factor for updated momentum
  lorentz = sqrtf(1.0f + px_updated * px_updated + py_updated * py_updated +
                  pz_updated * pz_updated / mass_square);

  // update position using calculated velocity
  float lorentz_mass = mass * lorentz; // Pre-compute to avoid recalculating
  float x_updated = particle.x + timestep * px_updated / lorentz_mass;
  float y_updated = particle.y + timestep * py_updated / lorentz_mass;
  float z_updated = particle.z + timestep * pz_updated / lorentz_mass;

  // update the particle with new location and momentum
  particle.update(x_updated, y_updated, z_updated, px_updated, py_updated,
                  pz_updated);
}

__global__ void update_particles(Particle *particles,
                                 ParticleHistory *particle_histories,
                                 float3 e_field, float3 b_field, float charge,
                                 float mass, float timestep,
                                 int total_particle_count, int latest_time) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < total_particle_count) {
    push_particle(particles[idx], e_field, b_field, timestep, charge, mass);
    particle_histories[idx].x[latest_time] = particles[idx].x;
    particle_histories[idx].y[latest_time] = particles[idx].y;
    particle_histories[idx].z[latest_time] = particles[idx].z;
    particle_histories[idx].px[latest_time] = particles[idx].px;
    particle_histories[idx].py[latest_time] = particles[idx].py;
    particle_histories[idx].pz[latest_time] = particles[idx].pz;
  }
}

void launch_update_particles(Particle *particles,
                             ParticleHistory *particle_histories,
                             float3 e_field, float3 b_field, float charge,
                             float mass, float timestep,
                             int total_particle_count, int iter) {
  const int threads_per_block = 256;
  const int blocks =
      (total_particle_count + threads_per_block - 1) / threads_per_block;
  update_particles<<<blocks, threads_per_block>>>(
      particles, particle_histories, e_field, b_field, charge, mass, timestep,
      total_particle_count, iter);
  hipDeviceSynchronize();
}

int main() {
  // Choose number particles and dimension to simulate
  const float3 num_of_particle = make_float3(4.0f, 4.0f, 4.0f);
  const float3 system_length = make_float3(4.0f, 4.0f, 4.0f);
  const int total_particle_count = static_cast<int>(
      num_of_particle.x * num_of_particle.y * num_of_particle.z);

  // Initialize E and B fields
  const float3 e_field = make_float3(0.5f, 0.5f, 0.5f);
  const float3 b_field = make_float3(0.75f, 0.75f, 0.75f);
  const float charge = 1.0f;
  const float mass = 1.0f;

  // choose arbitrary timesteps
  const float timestep = 0.025f;

  // Number of Boris pusher iteration run
  const int max_iter = 100;

  // Declare and initialize particles with fixed data
  Particle *particles = new Particle[total_particle_count];
  ParticleHistory *particle_histories =
      new ParticleHistory[total_particle_count];
  initialize_particles(particles, num_of_particle, total_particle_count,
                       system_length, ParticlePlacementType::UNIFORM);
  initialize_particle_history(particle_histories, total_particle_count,
                              max_iter);

  // // Print particle initialize
  // for (int i = 0; i < total_particle_count; i++)
  // {
  //     std::cout << "Particle " << i << ": " << particles[i].print() << "\n";
  // }

  // Allocate and copy particles to device 
  int particle_mem_size = 6 * sizeof(float);
  Particle *device_particles;
  hipMalloc(&device_particles, particle_mem_size);
  hipMemcpy(device_particles, particles, particle_mem_size,
             hipMemcpyHostToDevice);

  // Allocate and copy particle_histories to device 
  int; 
  ParticleHistory *device_particle_histories;

  for (int i = 0; i < max_iter; i++) {
    launch_update_particles(device_particles, device_particle_histories, e_field,
                            b_field, charge, mass, timestep,
                            total_particle_count, 0);
  }

  // Define, create, and start recording CUDA events
  // CUDAMEMCPY from device back to host
  // Sync event and check time taken
  // free everything

  // // Print particle
  // for (int i = 0; i < total_particle_count; i++)
  // {
  //     std::cout << "Particle " << i << ": " << particles[i].print() << "\n";
  // }
  return 0.0;
}
