#include <algorithm>
#include <hip/hip_runtime.h>
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime_api.h>
#include <hip/driver_types.h>
#include <iostream>
#include <math.h>
#include <string>
#include <vector_functions.h>

struct Particle {
  float x;
  float y;
  float z;

  float px;
  float py;
  float pz;

  // Default constructor
  Particle() : x(), y(), z(), px(), py(), pz() {}

  // Allow function to run on both host and kernel
  __host__ __device__
  void update(float x_pos, float y_pos, float z_pos, float x_momentum,
              float y_momentum, float z_momentum) {
    x = x_pos;
    y = y_pos;
    z = z_pos;
    px = x_momentum;
    py = y_momentum;
    pz = z_momentum;
  }

  std::string print() {
    return "x: " + std::to_string(x) + " y: " + std::to_string(y) +
           " z: " + std::to_string(z) + " px: " + std::to_string(px) +
           " py: " + std::to_string(py) + " pz: " + std::to_string(pz);
  }
};

struct ParticleHistory {
  float *x;
  float *y;
  float *z;
  float *px;
  float *py;
  float *pz;

  // Default constructor
  ParticleHistory()
      : x(nullptr), y(nullptr), z(nullptr), px(nullptr), py(nullptr),
        pz(nullptr) {}

  // Initialize the variable lists of size max_time
  ParticleHistory(int max_time) {
    x = new float[max_time];
    y = new float[max_time];
    z = new float[max_time];
    px = new float[max_time];
    py = new float[max_time];
    pz = new float[max_time];
  }

  std::string print(int time) {
    return "x: " + std::to_string(x[time]) + " y: " + std::to_string(y[time]) +
           " z: " + std::to_string(z[time]) +
           " px: " + std::to_string(px[time]) +
           " py: " + std::to_string(py[time]) +
           " pz: " + std::to_string(pz[time]);
  }

  ~ParticleHistory() {
    delete[] x;
    delete[] y;
    delete[] z;
    delete[] px;
    delete[] py;
    delete[] pz;
  }
};

enum class ParticlePlacementType {
  UNIFORM, // Evenly spaced integer placement for xyz coordinates
};

void initialize_particles(Particle *particles, float3 num_of_particles,
                          int total_particle_count, float3 system_length,
                          ParticlePlacementType placement_type) {
  switch (placement_type) {
  case ParticlePlacementType::UNIFORM:
    float deltax = system_length.x / num_of_particles.x;
    float deltay = system_length.y / num_of_particles.y;
    float deltaz = system_length.z / num_of_particles.z;
    for (int i = 0; i < total_particle_count; i++) {
      float corrected_index = static_cast<float>(i) + 0.5f;
      particles[i].update(deltax * corrected_index, deltay * corrected_index,
                          deltaz * corrected_index, 10, 10, 10);
    }
    break;
  }
}

void initialize_particle_history(ParticleHistory *particle_histories,
                                 int total_particle_count, int max_iter) {
  // Initialize for all particles
  for (int i = 0; i < total_particle_count; i++) {
    particle_histories[i] = ParticleHistory(max_iter);
  }
}

void cudaMalloc_particle_history(ParticleHistory *particle_histories,
                                 int total_particle_count, int max_iter) {
  // As this is going on the device, we must use cudaMalloc
  int floatarr_mem_size = max_iter * sizeof(float);
  for (int i = 0; i < total_particle_count; i++) {
    hipMalloc(&particle_histories[i].x, floatarr_mem_size);
    hipMalloc(&particle_histories[i].y, floatarr_mem_size);
    hipMalloc(&particle_histories[i].z, floatarr_mem_size);
    hipMalloc(&particle_histories[i].px, floatarr_mem_size);
    hipMalloc(&particle_histories[i].py, floatarr_mem_size);
    hipMalloc(&particle_histories[i].pz, floatarr_mem_size);
  }
}

__device__ void push_particle(Particle &particle, const float3 &e_field,
                              const float3 &b_field, const float timestep,
                              const float charge, float mass) {
  // Pre-compute reused calculations
  float mass_square = mass * mass;
  float timestep_charge = charge * timestep;

  // half-step momentum from electric field
  float px_half = particle.px + timestep_charge * e_field.x / 2.0f;
  float py_half = particle.py + timestep_charge * e_field.y / 2.0f;
  float pz_half = particle.pz + timestep_charge * e_field.z / 2.0f;

  // lorentz factor for half-step momentum
  float lorentz = sqrtf(1.0f + px_half * px_half + py_half * py_half +
                        pz_half * pz_half / mass_square);

  // rotation vector from magnetic field
  float lorentz_double = 2.0f * lorentz; // Pre-compute to avoid recalculating
  float tx = timestep_charge * b_field.x / lorentz_double;
  float ty = timestep_charge * b_field.y / lorentz_double;
  float tz = timestep_charge * b_field.z / lorentz_double;
  float t_mag_square = tx * tx + ty * ty + tz * tz;

  // cross product of half p and t
  float px_prime = px_half + (py_half * tz - pz_half * ty);
  float py_prime = py_half + (pz_half * tx - px_half * tz);
  float pz_prime = pz_half + (px_half * ty - py_half * tx);

  // update momentum with effect from boris rotation and electric field
  float denominator = 1 + t_mag_square;
  float px_updated = px_half +
                     2 * (py_prime * tz - pz_prime * ty) / denominator +
                     timestep_charge * e_field.x / 2.0f;
  float py_updated = py_half +
                     2 * (pz_prime * tx - px_prime * tz) / denominator +
                     timestep_charge * e_field.y / 2.0f;
  float pz_updated = pz_half +
                     2 * (px_prime * ty - py_prime * tx) / denominator +
                     timestep_charge * e_field.z / 2.0f;

  // lorentz factor for updated momentum
  lorentz = sqrtf(1.0f + px_updated * px_updated + py_updated * py_updated +
                  pz_updated * pz_updated / mass_square);

  // update position using calculated velocity
  float lorentz_mass = mass * lorentz; // Pre-compute to avoid recalculating
  float x_updated = particle.x + timestep * px_updated / lorentz_mass;
  float y_updated = particle.y + timestep * py_updated / lorentz_mass;
  float z_updated = particle.z + timestep * pz_updated / lorentz_mass;

  // update the particle with new location and momentum
  particle.update(x_updated, y_updated, z_updated, px_updated, py_updated,
                  pz_updated);
}

__global__ void update_particles(Particle *particles,
                                 ParticleHistory *particle_histories,
                                 float3 e_field, float3 b_field, float charge,
                                 float mass, float timestep,
                                 int total_particle_count, int latest_time) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < total_particle_count) {
    push_particle(particles[idx], e_field, b_field, timestep, charge, mass);
    particle_histories[idx].x[latest_time] = particles[idx].x;
    particle_histories[idx].y[latest_time] = particles[idx].y;
    particle_histories[idx].z[latest_time] = particles[idx].z;
    particle_histories[idx].px[latest_time] = particles[idx].px;
    particle_histories[idx].py[latest_time] = particles[idx].py;
    particle_histories[idx].pz[latest_time] = particles[idx].pz;
  }
}

void launch_update_particles(Particle *particles,
                             ParticleHistory *particle_histories,
                             float3 e_field, float3 b_field, float charge,
                             float mass, float timestep,
                             int total_particle_count, int iter) {
  const int threads_per_block = 256;
  const int blocks =
      (total_particle_count + threads_per_block - 1) / threads_per_block;
  update_particles<<<blocks, threads_per_block>>>(
      particles, particle_histories, e_field, b_field, charge, mass, timestep,
      total_particle_count, iter);
  hipDeviceSynchronize();
}

int main() {
  // Choose number particles and dimension to simulate
  const float3 num_of_particle = make_float3(4.0f, 4.0f, 4.0f);
  const float3 system_length = make_float3(4.0f, 4.0f, 4.0f);
  const int total_particle_count = static_cast<int>(
      num_of_particle.x * num_of_particle.y * num_of_particle.z);

  // Initialize E and B fields
  const float3 e_field = make_float3(0.5f, 0.5f, 0.5f);
  const float3 b_field = make_float3(0.75f, 0.75f, 0.75f);
  const float charge = 1.0f;
  const float mass = 1.0f;

  // choose arbitrary timesteps
  const float timestep = 0.025f;

  // Number of Boris pusher iteration run
  const int max_iter = 100;

  // Declare and initialize particles with fixed data
  Particle *particles = new Particle[total_particle_count];
  ParticleHistory *particle_histories =
      new ParticleHistory[total_particle_count];
  initialize_particles(particles, num_of_particle, total_particle_count,
                       system_length, ParticlePlacementType::UNIFORM);
  initialize_particle_history(particle_histories, total_particle_count,
                              max_iter);

  // Allocate and copy particles to device
  int particle_mem_size = total_particle_count * sizeof(Particle);
  Particle *device_particles;
  hipMalloc(&device_particles, particle_mem_size);
  hipMemcpy(device_particles, particles, particle_mem_size,
             hipMemcpyHostToDevice);

  // Allocate and copy particle_histories to device
  int particle_history_mem_size =
      total_particle_count * sizeof(ParticleHistory);
  ParticleHistory *device_particle_histories;
  hipMalloc(&device_particle_histories, particle_history_mem_size);
  cudaMalloc_particle_history(device_particle_histories, total_particle_count,
                              max_iter);
  hipMemcpy(device_particle_histories, particle_histories,
             particle_history_mem_size, hipMemcpyHostToDevice);

  // Create events to time the memcpy transfer
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);
  for (int i = 0; i < max_iter; i++) {
    launch_update_particles(device_particles, device_particle_histories,
                            e_field, b_field, charge, mass, timestep,
                            total_particle_count, i);
  }
  hipMemcpy(particle_histories, device_particle_histories,
             particle_history_mem_size, hipMemcpyDeviceToHost);

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);

  float time_elapsed = 0;
  hipEventElapsedTime(&time_elapsed, start, stop);

  std::cout << "Elapsed time: " << time_elapsed << "\n";

  // Destroy events
  hipEventDestroy(start);
  hipEventDestroy(stop);

  // print particle
  for (int i = 0; i < total_particle_count; i++) {
    std::cout << "particle " << i << ": "
              << particle_histories[i].print(max_iter - 1) << "\n";
  }

  return 0.0;
}
